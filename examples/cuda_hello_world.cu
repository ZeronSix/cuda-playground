
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void CudaHello() { printf("Hello World from GPU!\n"); }

int main() {
    CudaHello<<<1, 1>>>();
    hipError_t cuda_error = hipDeviceSynchronize();
    if (cuda_error != hipSuccess) {
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cuda_error));
    }

    return 0;
}
