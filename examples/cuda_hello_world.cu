#include "hip/hip_runtime.h"
#include <stdio.h>

#include <cuda_playground_common/cuda_assert.h>

__global__ void CudaHello() {
    printf("Hello World from GPU!\n");
}

int main() {
    CudaHello<<<1, 1>>>();
    CUDA_ASSERT(hipDeviceSynchronize());

    return 0;
}
